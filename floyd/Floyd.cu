#include "hip/hip_runtime.h"
#include <stdint.h>
#include <bits/stdc++.h>
#include <time.h>
#include "utils.h"

using namespace std;

int random_seed=1234;


__global__ void Floyd_Warshall_CUDA(int V, int i,unsigned long* CUDA_Matrix);

struct timespec start, endtime;

int main(int argc, char** argv){

    cout<<"Floyd Warshall's Algorithm: Sequential vs CUDA Comparison\n\n";
    int N = (int) atoi(argv[1]);

    int graph_size=N*N;
    unsigned long *Matrix=(unsigned long *)calloc(graph_size, sizeof(unsigned long));
    unsigned long *CPU_Matrix=(unsigned long *)calloc(graph_size, sizeof(unsigned long));
    unsigned long *GPU_Matrix=(unsigned long *)calloc(graph_size, sizeof(unsigned long));

    cout<<"N : "<<N<<endl;

    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            Matrix[i*N + j] = 0;
        }
    }

    generateRandomGraph(Matrix, N, random_seed);

    
    for(int i=0;i<graph_size;i++){
        CPU_Matrix[i]=Matrix[i];
    }
    // for(int i=0;i<N;i++){
    //     for(int j=0;j<N;j++){
    //         cout<< CPU_Matrix[i*N + j] << " ";
    //     }
    //     cout<<endl;
    // }
    // cout<<"After CPU\n";
    clock_gettime(CLOCK_MONOTONIC, &start);
    Floyd_Warshall_CPU(CPU_Matrix, N);
    clock_gettime(CLOCK_MONOTONIC, &endtime);
    double diff=0;
    // for(int i=0;i<N;i++){
    //     for(int j=0;j<N;j++){
    //         cout<< CPU_Matrix[i*N + j] << " ";
    //     }
    //     cout<<endl;
    // }
    diff=timetaken(start, endtime);;

    cout<<"Time taken for Floyd Warshall on CPU: "<<diff<<endl;

    for(int i=0;i<graph_size;i++){
        GPU_Matrix[i]=Matrix[i];
    }


    unsigned long* CUDA_Matrix;

    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMalloc((void**)&CUDA_Matrix,graph_size*sizeof(unsigned long));

    hipMemcpy(CUDA_Matrix, GPU_Matrix, graph_size*sizeof(unsigned long), hipMemcpyHostToDevice);
    int block_size = 512;
    dim3 dimGrid((N+block_size-1)/block_size,N);   

    for(int i=0;i<N;i++){
       Floyd_Warshall_CUDA<<<dimGrid,block_size>>>(N, i, CUDA_Matrix);
       hipDeviceSynchronize();
    }
    
    hipMemcpy(GPU_Matrix, CUDA_Matrix, graph_size*sizeof(unsigned long), hipMemcpyDeviceToHost);
    // cout<<endl;
    // for(int i=0;i<N;i++){
    //     for(int j=0;j<N;j++){
    //         cout<< GPU_Matrix[i*N + j] << " ";
    //     }
    //     cout<<endl;
    // }
    
    clock_gettime(CLOCK_MONOTONIC, &endtime);
    double diffGPU=timetaken(start, endtime);;
    
    cout<<"Time taken for Floyd Warshall on CUDA: "<<diffGPU<<endl;

    int match=graph_size;
    for(int i=0;i<graph_size;i++){
        match -= int(CPU_Matrix[i]==GPU_Matrix[i]);
    }
    if(match==0){
       cout<<"The sequential and CUDA outputs match!\n";
    }

    free(Matrix);
    free(CPU_Matrix);
    free(GPU_Matrix);
    hipFree(CUDA_Matrix);

}

__global__ void Floyd_Warshall_CUDA(int n, int k,unsigned long* CUDA_Matrix){

    int i=blockIdx.x*blockDim.x +threadIdx.x;
    if(i>=n) return;
    __shared__  int min_distance;
 

    if(threadIdx.x==0){
        min_distance=CUDA_Matrix[n*blockIdx.y+k];
    }

    __syncthreads();

    int idx=n*blockIdx.y + i;
    unsigned long currDistance=CUDA_Matrix[k*n+i];
    unsigned long total_distance=min_distance+currDistance;
    if (CUDA_Matrix[idx] > total_distance) CUDA_Matrix[idx] = total_distance;
}



