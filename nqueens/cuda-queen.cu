#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>


/*** TODO: insert the declaration of the kernel function below this line ***/

__global__ void isValid(int *board, int total, int N, bool *results);

/**** end of the kernel declaration ***/


int main(int argc, char *argv[]){

	int n = 0; 
	int i;  
	int *cpu_boards; 
	int *gpu_boards;
	bool *cpu_results;
	bool *gpu_results;
	clock_t start, end; 
	hipSetDevice(1);
	
	if(argc != 2){
		printf("usage:  ./vectorprog n\n");
		printf("n = number of elements in each vector\n");
		exit(1);
		}
		
	n = atoi(argv[1]);
	
	int max_iter = 1;
    for (int i = 0; i < n; i++)
      max_iter *= n;

	
	if( !(cpu_boards = (int *)malloc(pow(n,n)*sizeof(int))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}
	if( !(cpu_results = (bool *)malloc(pow(n,n)*sizeof(bool))) )
	{
	   printf("Error allocating array a\n");
	   exit(1);
	}

	start = clock();
	
	size_t space = pow(n,n)*sizeof(int);
	hipHostMalloc(&gpu_boards, space, hipHostMallocDefault);

	size_t space2 = pow(n,n)*sizeof(bool);
	hipHostMalloc(&gpu_results, space2, hipHostMallocDefault);

	long iter = 0;
	int idx;
	int number;
	for(iter = 0;iter < max_iter; iter++){
		idx = iter;
		number = 0;
		for(int i=0;i<n;i++){
			number *= 10;
			number += idx % n;
			idx /= n;
		}
		cpu_boards[iter] = number;
		cpu_results[iter] = false;
	}

	hipMemcpy(gpu_boards, cpu_boards, space, hipMemcpyHostToDevice);
	hipMemcpy(gpu_results, cpu_results, space2, hipMemcpyHostToDevice);
	
	int NUM_THREADS = 512;
	int NUM_BLOCKS = max_iter/NUM_THREADS+1;

	
	isValid<<< NUM_BLOCKS, NUM_THREADS >>>(gpu_boards, max_iter, n, gpu_results);

	hipMemcpy(cpu_results, gpu_results, space2, hipMemcpyDeviceToHost);

	hipHostFree(gpu_results); hipHostFree(gpu_boards); 
	
	end = clock();
	printf("Total time taken by the GPU part = %lf\n", (double)(end - start) / CLOCKS_PER_SEC);

	int counter = 0;
	for(i = 0; i < max_iter; i++)
	  if( cpu_results[i]) 
		counter++;

	printf("Final count: %d\n",counter);
		
	free(cpu_results); free(cpu_boards); 

	return 0;
}


/**** TODO: Write the kernel itself below this line *****/
__global__ void isValid(int *board, int total, int N, bool *results){
	int index = blockDim.x * blockIdx.x + threadIdx.x;
    if(index >= total)
        return;
	int number = board[index];
    int errors = 0;
	int ithDigit;
	int jthDigit;
    for(int i=0;i<N;i++){
		ithDigit = number / pow(10.0,(double)i);
		ithDigit = ithDigit % 10;
        for(int j=0;j<N;j++){
			jthDigit = number / pow(10.0,(double)j);
			jthDigit = jthDigit % 10;
            if(i<j && ithDigit == jthDigit) errors++;
            if (i < j && (ithDigit - jthDigit == i - j || ithDigit - jthDigit == j - i)) errors++;
        }
    }
	if(errors > 0)
        results[index] = false;
    else
        results[index] = true;
    
}
