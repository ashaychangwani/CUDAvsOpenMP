#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include "utils.h"

#define square(X) X*X
#define THREADS_PER_BLOCK 512

/*****************************************************************/

/*** Kernel Definitions ***/
__global__ void find_cluster(int, int, Pixel*, const Cluster* __restrict__);
__global__ void recenter1(int, int, Pixel*, uint4*);
__global__ void recenter2(Cluster*, uint4*, bool*);
/**** end of the kernel declaration ***/

/*****************************************************************/

int main(int argc, char * argv[]) {
	
	if(argc != 4){
        fprintf(stderr, "usage: kmeans_sequential <IN_PATH> <OUT_PATH> <K_CLUSTERS> \n");
        exit(1);
    }

	const char *inPath, *outPath;
	inPath = argv[1]; outPath = argv[2];
	int K_clusters = atoi(argv[3]);
	unsigned int height, width, channels;
	
	unsigned char* image;

	if (read_png(inPath, &image, height, width, channels)!= 0) {
		exit(1);
	}
	if (channels !=3){
		printf("Three channel PNG only supported as of now");
	}

	int n_pixels = height * width;
	Pixel* pixels = (Pixel*)calloc(n_pixels, sizeof(Pixel));
	int i=0;
	while(i<n_pixels){
		pixels[i].x = image[3*i+ 0];
		pixels[i].y = image[3*i+ 1];
		pixels[i].z = image[3*i+ 2];
		pixels[i].cluster = -1;
		i++;
	}

	Cluster* clusters = (Cluster*)calloc(K_clusters, sizeof(Cluster));
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> uniform(0, n_pixels - 1);
	i=0;
	//Initialize Cluster and Assign a Random Pixel to the Cluster
	while (i<K_clusters){
		Pixel *pixel = &pixels[uniform(gen)];
		clusters[i++] = Cluster(pixel->x, pixel->y, pixel->z, 0, (int*)calloc(n_pixels, sizeof(int)));
	}
	//Define Blocks and Threads per block
    dim3 numBlocks((n_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 threadsPerBlock(THREADS_PER_BLOCK);

	// allocate device memory
	Pixel* d_pixels;
	Cluster* d_cluster;
	size_t sz_pixel= sizeof(Pixel), sz_cluster = sizeof(Cluster);
	uint4* d_sum;	//To store sum and count
	bool* d_converged;

	hipMalloc((void**)&d_pixels, n_pixels * sz_pixel);
	hipMalloc((void**)&d_cluster, K_clusters * sz_cluster);
	hipMalloc((void**)&d_sum, sizeof(uint4));
	hipMalloc((void**)&d_converged, sizeof(bool));

	if(!d_pixels && !d_cluster && !d_sum && !d_converged){
        printf("cannot allocate array\n");
        exit(1);
    }

	hipMemcpy(d_pixels, pixels, n_pixels * sz_pixel, hipMemcpyHostToDevice);
	hipMemcpy(d_cluster, clusters, K_clusters * sz_cluster, hipMemcpyHostToDevice);


	bool thread_converged = true;
	bool converged;
	do{
		find_cluster<<<numBlocks , threadsPerBlock>>>(n_pixels, K_clusters, d_pixels, d_cluster);
		thread_converged = true;
		for (int i = 0; i < K_clusters; ++i) {
			hipMemset(d_sum, 0, 4 * sizeof(int));

			recenter1<<<numBlocks , threadsPerBlock>>>(n_pixels, i, d_pixels, d_sum);
			recenter2<<<1, 1>>>(&d_cluster[i], d_sum, d_converged);

			hipMemcpy(&converged, d_converged, sizeof(bool), hipMemcpyDeviceToHost);

			thread_converged &= converged;
		}

	}while (!thread_converged);

	// copy device memory back to host
	hipMemcpy(pixels, d_pixels, n_pixels * sz_pixel, hipMemcpyDeviceToHost);
	hipMemcpy(clusters, d_cluster, K_clusters * sz_cluster, hipMemcpyDeviceToHost);

	// free device memory
	hipFree(d_pixels);
	hipFree(d_cluster);
	hipFree(d_sum);

	int idx = 0;
	while(idx < n_pixels){
		Cluster* cluster = &clusters[pixels[idx].cluster];
		image[3*idx] = cluster->x;
		image[3*idx+1] = cluster->y;
		image[3*idx+2] = cluster->z;
		idx++;
	}

	if ((write_png(outPath, image, height, width, 3)) != 0) {
		printf("fail to write output png file\n");
		exit(1);
	}

	delete[] clusters;
	delete[] pixels;

	return 0;
}


__global__ void find_cluster(int n_pixels, int K_clusters, Pixel* pixels, const Cluster* __restrict__ clusters) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < n_pixels){
		Pixel pixel = pixels[idx];
		int min = INT_MAX, min_cluster, dist, j=0;

		while(j<K_clusters){
			dist = square((pixel.x - clusters[j].x))+ square((pixel.y - clusters[j].y)) + square((pixel.z - clusters[j].z));
			if (dist < min) {
				min = dist;
				min_cluster = j;
			}
			j++;
		}
		pixels[idx].cluster = min_cluster;
	}
}

__global__ void recenter1(int n_pixels, int cluster, Pixel* pixels, uint4* sumc) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < n_pixels){
		Pixel pixel = pixels[idx];
		if (pixel.cluster == cluster) {
			atomicAdd(&sumc->x, pixel.x);
			atomicAdd(&sumc->y, pixel.y);
			atomicAdd(&sumc->z, pixel.z);
			atomicAdd(&sumc->w, 1);
		}
	}

}

__global__ void recenter2(Cluster* cluster, uint4* sumc, bool* converged) {
	uint32_t points = sumc->w ;
	*converged = false;
	if (points > 0) {
		Cluster copy = *cluster;
		cluster->x = (sumc->x) / (points);
		cluster->y = (sumc->y) / (points);
		cluster->z = (sumc->z) / (points);
		if (cluster->x == copy.x && 
			cluster->y == copy.y && 
			cluster->z == copy.z)	*converged=true;
	}
}
